#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <string.h>
#include "hip/hip_runtime.h"

#define BLOCK_SIZE 32

typedef unsigned char uchar;
typedef unsigned int uint;
typedef unsigned long ulong;

__global__ void consolidateHistogram(ulong*blockHistograms,ulong* cudaHistogram,uint numBlocks) {
    int tid = threadIdx.x;

    for (uint j=0;j<numBlocks;j++) {
        cudaHistogram[tid]+=blockHistograms[j*256+tid];
    }    
}

__global__ void calculateHistogram(uchar* data,ulong* blockHistograms,ulong N) {
    int tid = threadIdx.x;
    uint x = threadIdx.x+blockIdx.x*blockDim.x;

    if (x<N) {
        __shared__ ulong sHistogram[BLOCK_SIZE][256];
        for (int i=0;i<256;i++)
            sHistogram[tid][i] = 0;
        __syncthreads();

        uint index = x;
        for (int i=0;i<256;i++) {
            if (index>=N)
                break;
            sHistogram[tid][data[index]]++;
            index+=BLOCK_SIZE;
        }

        __syncthreads();

        int blockSize = 256/32;
        int startIndex = tid*blockSize;
        int endIndex = startIndex+blockSize;
        if (tid==BLOCK_SIZE-1)
            endIndex+= 256%32;
        
        uint offset = 256*blockIdx.x;
        for (int i=startIndex;i<endIndex;i++) {
            for (int j=0;j<BLOCK_SIZE;j++)
                blockHistograms[offset+i]+=sHistogram[j][i];
        }
    }
}

void validate(ulong* arr1,ulong* arr2) {
    uchar incorrectCount = 0;
    for (int i=0;i<256;i++) {
        if (arr1[i]!=arr2[i])
            incorrectCount++;
    }

    if (incorrectCount==0)
        printf("Passed validation\n");
    else
        printf("%u bins were incorrect\n",incorrectCount);
}

int main(int argc,char** argv) {
    ulong DATA_SIZE  = 1 << 28;
    srand(2019);
    uchar* data = (uchar*) malloc(DATA_SIZE);
    
    for (uint i=0;i<DATA_SIZE;i++) {
        data[i] = rand()%256;
    }

    double serialDuration = -1;
    double start = omp_get_wtime();
    ulong serialHistogram[256];
    memset(serialHistogram,0,sizeof(ulong)*256);
    for (uint i=0;i<DATA_SIZE;i++ ) {
        serialHistogram[data[i]]++;
    }
    double end = omp_get_wtime();
    serialDuration = end - start;
    printf("Serial operation took %.5f seconds to run.Speed up -\n",serialDuration);

    double parallelDuration = -1;
    start = omp_get_wtime();
    ulong parallelHistogram[256];
    memset(parallelHistogram,0,sizeof(ulong)*256);

    #pragma omp parallel
    {
        ulong partialHistogram[256];
        memset(partialHistogram,0,sizeof(ulong)*256);
        int numThreads = omp_get_num_threads();
        int tid = omp_get_thread_num();
        ulong blockSize = DATA_SIZE/numThreads;
        ulong startIndex = blockSize*tid;
        ulong endIndex = startIndex+blockSize;

        if (tid == numThreads-1)
            endIndex+= DATA_SIZE%numThreads;
        
        for (ulong i=startIndex;i<endIndex;i++) {
            partialHistogram[data[i]]++;
        }

        for (int i=0;i<256;i++) {
            #pragma omp critical
            {
                parallelHistogram[i]+=partialHistogram[i];
            }
        }
    }

    end = omp_get_wtime();
    parallelDuration = end-start;
    printf("Parallel operation took %.5f seconds to run. Speed up %.1f\n",parallelDuration,serialDuration/parallelDuration);

    validate(serialHistogram,parallelHistogram);

    double cudaDuration = -1;
    start = omp_get_wtime();
    ulong* cudaHistogram = (ulong*) malloc(sizeof(ulong)*256);
    memset(cudaHistogram,0,sizeof(ulong)*256);
    uint numBlocks = (DATA_SIZE+32*256)/ (32*256);
    ulong* blockHistograms = (ulong*) malloc(sizeof(ulong)*numBlocks*256);
    for (int i=0;i<numBlocks*256;i++)
        blockHistograms[i] = 0;

    struct CudaContext cudaContext;
    cudaContext.init();

    ulong* deviceBlockHistogram = (ulong*)cudaContext.cudaInOut((void*) blockHistograms,sizeof(ulong)*numBlocks*256);

    calculateHistogram<<<numBlocks,32>>>(
        (uchar*)cudaContext.cudaIn((void*) data,sizeof(uchar)*DATA_SIZE),
        deviceBlockHistogram,
        DATA_SIZE);

    consolidateHistogram<<<1,256>>>(
        deviceBlockHistogram,
        (ulong*) cudaContext.cudaInOut((void*) cudaHistogram,sizeof(ulong)*256),
        numBlocks);

    cudaContext.synchronize((void*)cudaHistogram);

    end = omp_get_wtime();
    cudaDuration = end-start;
    printf("Cuda operation took %.5f seconds to run. Speed up %.1f\n",cudaDuration,serialDuration/cudaDuration);

    validate(serialHistogram,cudaHistogram);
    free(data);
    cudaContext.dispose();

    return 0;
}