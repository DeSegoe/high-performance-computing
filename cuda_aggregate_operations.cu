#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include "hip/hip_runtime.h"

#define DATA_SIZE 1 << 28

typedef unsigned char uchar;
typedef unsigned int uint;
typedef unsigned long ulong;

__global__ void aggregator(uchar* globalData,ulong *sum,uint N) {
    uint x = threadIdx.x + blockIdx.x*blockDim.x;

    if (x<N && threadIdx.x<N) {
        __shared__ uint sharedData[1024];
        
        sharedData[threadIdx.x] = globalData[x];

        __syncthreads();

        int halfBlock = blockDim.x/2;

        for (int s=1;s<halfBlock;s++) {
            int index = 2*s*threadIdx.x;
            if (index<blockDim.x)
                sharedData[index]+= sharedData[index+s];

            __syncthreads();
        }

        if (threadIdx.x==0) {
            int number = 0;
            atomicAdd(&number,1);
            sum[0]+= sharedData[0];
        }
    }
}

int main(int argc,char** argv) {
    srand(2019);
    uchar* data = (uchar*) malloc(DATA_SIZE);
    
    for (int i=0;i<DATA_SIZE;i++) {
        data[i] = rand()%256;
    }

    ulong serialCount = 0;
    double start = omp_get_wtime();
    for (uint i=0;i<DATA_SIZE;i++ ) {
        serialCount+=data[i];
    }
    double end = omp_get_wtime();
    printf("Serial operation took %.5f seconds to run. The total is %u, Speed up -\n",end-start,serialCount);

    ulong parallelCount = 0;
    struct CudaContext cudaContext;
    cudaContext.init();
    start = omp_get_wtime();
    const int numberOfThreads = 1024;
    const int numberOfBlocks = cudaContext.getBlocks(DATA_SIZE);
    aggregator<<<numberOfBlocks,numberOfThreads>>>(
       (uchar*) cudaContext.cudaIn((void*) data,DATA_SIZE),
       (ulong*) cudaContext.cudaInOut((void*) &parallelCount,sizeof(ulong)),
       DATA_SIZE);
    //cudaContext.synchronize();
    end = omp_get_wtime();
    printf("Parallel operation took %.5f seconds to run. The total is %u, Speed up -\n",end-start,parallelCount);
    
    cudaContext.dispose();
    free(data);

    printf("Finished");

    return 0;
}